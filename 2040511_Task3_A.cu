#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime_api.h>

#include <time.h>



//compile with: nvcc 2040511_Task3_A.cu -o 2040511_Task3_A



__device__ int is_a_match(char *attempt) {

	char plain_password1[] = "AR29";

	char plain_password2[] = "RR70";

	char plain_password3[] = "PP90";

	char plain_password4[] = "KO78";





	char *s = attempt;

	char *u = attempt;

	char *a = attempt;

	char *y = attempt;

	char *p1 = plain_password1;

	char *p2 = plain_password2;

	char *p3 = plain_password3;

	char *p4 = plain_password4;



	while(*s == *p1) {

		if(*s == '\0')

		{

			printf("Password: %s\n",plain_password1);

			break;

		}



		s++;

		p1++;

	}



	while(*u == *p2) {

		if(*u == '\0')

		{

			printf("Password: %s\n",plain_password2);

			break;

		}



		u++;

		p2++;

	}



	while(*a == *p3) {

		if(*a == '\0')

		{

			printf("Password: %s\n",plain_password3);

			break;

		}



		a++;

		p3++;

	}



	while(*y == *p4) {

		if(*y == '\0')

		{

			printf("Password: %s\n",plain_password4);

			return 1;

		}



		y++;

		p4++;

	}

	return 0;



}



__global__ void  kernel() {

	char i1,i2;



	char password[5];

	password[4] = '\0';



	int i = blockIdx.x+65;

	int j = threadIdx.x+65;

	char firstMatch = i;

	char secondMatch = j;



	password[0] = firstMatch;

	password[1] = secondMatch;

	for(i1='0'; i1<='9'; i1++){

		for(i2='0'; i2<='9'; i2++){



					password[2] = i1;

					password[3] = i2;



					if(is_a_match(password)) {

					}

					else {



					}

				}

			}

		}




int time_difference(struct timespec *start,

	struct timespec *finish,

	long long int *difference) {

	long long int ds =  finish->tv_sec - start->tv_sec;

	long long int dn =  finish->tv_nsec - start->tv_nsec;

	if(dn < 0 ) {

		ds--;

		dn += 1000000000;

	}   

	*difference = ds * 1000000000 + dn;

	return !(*difference > 0);

}





int main() {



	struct  timespec start, finish;

	long long int time_elapsed;

	clock_gettime(CLOCK_MONOTONIC, &start);



	kernel <<<26,26>>>();

	hipDeviceSynchronize();



	clock_gettime(CLOCK_MONOTONIC, &finish);

	time_difference(&start, &finish, &time_elapsed);

	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));



	return 0;

}


